#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// most of this is taken directly from llm.c.
// https://github.com/karpathy/llm.c/blob/master/dev/cuda/attention_forward.cu

// Kernels for attention forward pass.

// permute kernel for qkv input
__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void scale_kernel(float* inp, float scale, int B, int NH, int T) {
    // scales the pre-softmax attention scores by scale
    // we don't want causal attention, so we don't set any values to infinity
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * T * T) {
        inp[idx] *= scale;
    }
}

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__global__ void softmax_forward_kernel4(float* out, const float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    const float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        // subtract max for numerical stability
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}

__global__ void unpermute_kernel(const float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}


void attention_forward1
(
    hipblasHandle_t cublas_handle,
    float* out, float* qkvr, float* preatt, float* att,
    float* inp,
    int B, int T, int C, int NH,
    const int block_size
) {
    // note: resure inp as a scratch buffer because it is not used in backward
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            T, T, HS,
                            &alpha,
                            k, HS, T * HS,
                            q, HS, T * HS,
                            &beta,
                            preatt, T, T * T,
                            B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0f / sqrtf(HS);
    total_threads = B * NH * T * T;
    num_blocks = ceil_div(total_threads, block_size);
    scale_kernel<<<num_blocks, block_size>>>(preatt, scale, B, NH, T);

    // softmax. preatt is (B, NH, T, T) but we view it as (B * NH * T, T) and use the softmax kernel
    int softmax_block_size = 256;
    int grid_size = B * NH * T;
    size_t shared_mem_size = 2 * softmax_block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, softmax_block_size, shared_mem_size>>>(att, preatt, B * NH * T, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    float* vaccum = inp;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, T, T,
                            &alpha,
                            v, HS, T * HS,
                            att, T, T * T,
                            &beta,
                            vaccum, HS, T * HS,
                            B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
}

__global__ void permute_kernel_backward(float* dinp,
                                        const float* dq, const float* dk, const float* dv,
                                        int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        dinp[inp_idx] = dq[idx];
        dinp[inp_idx + NH * d] = dk[idx];
        dinp[inp_idx + 2 * (NH * d)] = dv[idx];
    }
}

__global__ void unpermute_kernel_backward(float* dinp, const float *dout, int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        dinp[idx] = dout[other_idx];
    }
}

// slight edit from the softmax_autoregressive_backward_kernel from train_gpt2_fp32
// our attention is no longer autoregressive
__global__ void softmax_backward_kernel(float* dpreatt, const float* datt, const float* att,
                                                       int B, int T, int C, float scale) {
    constexpr const int BlockSize = 256;
    constexpr int T_per_block = 4;
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float block_acc[32];

    int idx = blockIdx.y;
    // go through blocks in reverse order, so the slowest block starts first
    int t0 = T - 1 - T_per_block*blockIdx.x;

    att += idx * T * T;
    datt += idx * T * T;
    dpreatt += idx * T * T;

    if (warp.meta_group_rank() == 0) {
        block_acc[warp.thread_rank()] = 0;
    }

    for(int to = 0; to < T_per_block; ++to) {
        int t = t0 - to;
        if(t < 0) return;
        const float* att_bth = att + t * T;
        const float* datt_bth = datt + t * T;
        float* dpreatt_bth = dpreatt + t * T;

        float local_sum = 0.0f;
        for (int t2 = block.thread_rank(); t2 < T; t2 += BlockSize) {
            local_sum += att_bth[t2] * datt_bth[t2];
        }

        block_acc[warp.meta_group_rank()] = cg::reduce(warp, local_sum, cg::plus<float>{});
        block.sync();
        local_sum = cg::reduce(warp, block_acc[warp.thread_rank()], cg::plus<float>{});

        for (int t3 = block.thread_rank(); t3 < T; t3 += BlockSize) {
            // don't touch the cache. Some parts will still be here from the previous loop, and
            // we want to exploit those.
            float acc = __ldcs(att_bth + t3) * (__ldcs(datt_bth + t3) - local_sum);
            __stcs(dpreatt_bth + t3, scale * acc);
        }
    }
}

// the sequence of transformations in this compound op is:
// inp (B,T,3C) -> qkvr (B,T,3C) -> preatt (B,NH,T,T) -> att (B,NH,T,T) -> vaccum (B,T,C) -> out (B,T,C)
void attention_backward
(
    hipblasHandle_t cublas_handle,
    float* dinp, float* dqkvr, float* dpreatt, float* datt, float* scratch,
    const float* dout,
    const float* qkvr, const float* att,
    int B, int T, int C, int NH
) {
    const int block_size = 256;
    int HS = C / NH; // head size
    const float one = 1.0f;
    const float zero = 0.0f; // note beta = 1.0f so that we accumulate gradients (+=)
    // unpack convenience pointers into q, k, v
    const float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    float *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;
    // backward through the unpermute operation
    int num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    cudaCheck(hipGetLastError());
    // backward into datt
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &one, v, HS, T * HS, scratch, HS, T * HS, &zero, datt, T, T * T, B * NH));
    // backward into dv
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, scratch, HS, T * HS, att, T, T * T, &zero, dv, HS, T * HS, B * NH));
    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_backward_kernel<<<dim3(T / 4, B * NH), 256>>>(dpreatt, datt, att, B, T, C, scale);
    cudaCheck(hipGetLastError());
    // backward into q
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &one, k, HS, T * HS, dpreatt, T, T * T, &zero, dq, HS, T * HS, B * NH));
    // backward into k
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, q, HS, T * HS, dpreatt, T, T * T, &zero, dk, HS, T * HS, B * NH));
    // backward into inp
    num_blocks = ceil_div(B * NH * T * HS, block_size);
    permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}


#ifndef LINKING
int main(int argc, char** argv) {
    int B = 4;
    int T = 1024;
    int C = 256;
    int HS = 32;
    int NH = C / HS;

    // setup cublas
    hipblasHandle_t cublas_handle;
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH));

    // create host memory to load data
    float* qkv = (float*)malloc(B * T * 3 * C * sizeof(float));
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* dout = (float*)malloc(B * T * C * sizeof(float));
    float* dqkv = (float*)malloc(B * T * 3 * C * sizeof(float));

    // read saved output
    FILE * file = fopen("attention.bin", "rb");
    if (!file) {
        perror("Failed to load data");
        return -1;
    }
    freadCheck(qkv, sizeof(float), B * T * 3 * C, file);
    freadCheck(out, sizeof(float), B * T * C, file);
    freadCheck(dout, sizeof(float), B * T * C, file);
    freadCheck(dqkv, sizeof(float), B * T * 3 * C, file);
    fclose(file);

    // allocate device memory
    float *d_inp, *d_qkvr, *d_out, *d_preatt, *d_att, *d_dout, *d_dinp, *d_dqkvr, *d_dpreatt, *d_datt, *d_scratch;
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, qkv, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&d_qkvr, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemset(d_qkvr, 0, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMemset(d_out, 0, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_preatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMemset(d_preatt, 0, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMemset(d_att, 0, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_dout, dout, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&d_dinp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemset(d_dinp, 0, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dqkvr, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemset(d_dqkvr, 0, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dpreatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMemset(d_dpreatt, 0, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_datt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMemset(d_datt, 0, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_scratch, B * T * C * sizeof(float)));
    cudaCheck(hipMemset(d_scratch, 0, B * T * C * sizeof(float)));

    // forward pass
    int block_sizes[] = {256, 512};
    printf("Checking forward pass\n");
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("\nBlock size: %d\n", block_size);
        attention_forward1(cublas_handle, d_out, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, block_size);
        validate_result(d_out, out, "out", B * T * C);
        // reset d_inp, which is used as a buffer
        cudaCheck(hipMemcpy(d_inp, qkv, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));
    }

    printf("\nChecking backward pass\n");
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("\nBlock size: %d\n", block_size);
        attention_backward(cublas_handle, d_dinp, d_dqkvr, d_dpreatt, d_datt, d_scratch, d_dout, d_qkvr, d_att, B, T, C, NH);
        validate_result(d_dinp, dqkv, "dinp", B * T * 3 * C);
    }

    printf("\nAll results match. Starting benchmarks.\n\n");
    printf("Forward pass benchmarks:\n");
    int repeat_times = 100;
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j ++) {
        int block_size = block_sizes[j];
        float elapsed_time = benchmark_kernel(
            repeat_times, attention_forward1,
            cublas_handle, d_out, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, block_size
        );
        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }
    printf("\nBackward pass benchmarks:\n");
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j ++) {
        int block_size = block_sizes[j];
        float elapsed_time = benchmark_kernel(
            repeat_times, attention_backward,
            cublas_handle, d_dinp, d_dqkvr, d_dpreatt, d_datt, d_scratch, d_dout, d_qkvr, d_att, B, T, C, NH
        );
        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(qkv);
    free(out);
    free(dout);
    free(dqkv);
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_qkvr));
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_preatt));
    cudaCheck(hipFree(d_att));
    cudaCheck(hipFree(d_dout));
    cudaCheck(hipFree(d_dinp));
    cudaCheck(hipFree(d_dqkvr));
    cudaCheck(hipFree(d_dpreatt));
    cudaCheck(hipFree(d_datt));
    cudaCheck(hipFree(d_scratch));

    // destroy cublas handle
    cublasCheck(hipblasDestroy(cublas_handle));
}
#endif