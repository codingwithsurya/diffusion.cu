#include "common.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// ----------------------------------------------------------------------------
// cuBLAS setup

hipblasHandle_t cublas_handle;

// utils below taken from llm.c

// ----------------------------------------------------------------------------
// fread convenience utils, with nice handling of error checking using macros
// simple replace fopen, fread, fclose, fseek
// with fopenCheck, freadCheck, fcloseCheck, fseekCheck

FILE *fopen_check(const char *path, const char *mode, const char *file, int line) {
    FILE *fp = fopen(path, mode);
    if (fp == NULL) {
        fprintf(stderr, "Error: Failed to open file '%s' at %s:%d\n", path, file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Path: %s\n", path);
        fprintf(stderr, "  Mode: %s\n", mode);
        fprintf(stderr, "---> HINT: try to re-run the associated python file\n");
        exit(EXIT_FAILURE);
    }
    return fp;
}

#define fopenCheck(path, mode) fopen_check(path, mode, __FILE__, __LINE__)

void fread_check(void *ptr, size_t size, size_t nmemb, FILE *stream, const char *file, int line) {
    size_t result = fread(ptr, size, nmemb, stream);
    if (result != nmemb) {
        if (feof(stream)) {
            fprintf(stderr, "Error: Unexpected end of file at %s:%d\n", file, line);
        } else if (ferror(stream)) {
            fprintf(stderr, "Error: File read error at %s:%d\n", file, line);
        } else {
            fprintf(stderr, "Error: Partial read at %s:%d. Expected %zu elements, read %zu\n",
                    file, line, nmemb, result);
        }
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Expected elements: %zu\n", nmemb);
        fprintf(stderr, "  Read elements: %zu\n", result);
        exit(EXIT_FAILURE);
    }
}

#define freadCheck(ptr, size, nmemb, stream) fread_check(ptr, size, nmemb, stream, __FILE__, __LINE__)

void fclose_check(FILE *fp, const char *file, int line) {
    if (fclose(fp) != 0) {
        fprintf(stderr, "Error: Failed to close file at %s:%d\n", file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        exit(EXIT_FAILURE);
    }
}

#define fcloseCheck(fp) fclose_check(fp, __FILE__, __LINE__)

void fseek_check(FILE *fp, long off, int whence, const char *file, int line) {
    if (fseek(fp, off, whence) != 0) {
        fprintf(stderr, "Error: Failed to seek in file at %s:%d\n", file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  Offset: %ld\n", off);
        fprintf(stderr, "  Whence: %d\n", whence);
        fprintf(stderr, "  File:   %s\n", file);
        fprintf(stderr, "  Line:   %d\n", line);
        exit(EXIT_FAILURE);
    }
}

#define fseekCheck(fp, off, whence) fseek_check(fp, off, whence, __FILE__, __LINE__)

// ----------------------------------------------------------------------------
// malloc error-handling wrapper util

void *malloc_check(size_t size, const char *file, int line) {
    void *ptr = malloc(size);
    if (ptr == NULL) {
        fprintf(stderr, "Error: Memory allocation failed at %s:%d\n", file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Size: %zu bytes\n", size);
        exit(EXIT_FAILURE);
    }
    return ptr;
}

#define mallocCheck(size) malloc_check(size, __FILE__, __LINE__)
